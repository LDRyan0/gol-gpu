#include "hip/hip_runtime.h"
#include "common.h"

#define INCLUDE_CPU_VERSION
#include "game_of_life.c"

void __cuda_check_error(hipError_t err, const char *file, int line){
	if(err != hipSuccess){
        fprintf(stderr, "CUDA error (%s:%d): %s\n", file, line, hipGetErrorString(err));
        exit(1);
    }
}

#define CUDA_CHECK_ERROR(X)({\
	__cuda_check_error((X), __FILE__, __LINE__);\
})

#define MALLOC_CHECK_ERROR(X)({\
    if ((X) == 0){\
        fprintf(stderr, "Malloc error (%s:%d): %i\n", __FILE__, __LINE__, (X));\
        exit(1);\
    }\
})

__global__ void gpu_game_of_life_step(int *current_grid, int *next_grid, int n, int m){
    int neighbours;
    int n_i[8], n_j[8];

    // get the unique index (turn into i, j coordinates) of the thread to operate
    // on the n x m matrix
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int i = idx / m;
    unsigned int j = idx % m;

    // count the number of neighbours, clockwise around the current cell.
    neighbours = 0;
    n_i[0] = i - 1; n_j[0] = j - 1;
    n_i[1] = i - 1; n_j[1] = j;
    n_i[2] = i - 1; n_j[2] = j + 1;
    n_i[3] = i;     n_j[3] = j + 1;
    n_i[4] = i + 1; n_j[4] = j + 1;
    n_i[5] = i + 1; n_j[5] = j;
    n_i[6] = i + 1; n_j[6] = j - 1;
    n_i[7] = i;     n_j[7] = j - 1;

    if(n_i[0] >= 0 && n_j[0] >= 0 && current_grid[n_i[0] * m + n_j[0]] == ALIVE) neighbours++;
    if(n_i[1] >= 0 && current_grid[n_i[1] * m + n_j[1]] == ALIVE) neighbours++;
    if(n_i[2] >= 0 && n_j[2] < m && current_grid[n_i[2] * m + n_j[2]] == ALIVE) neighbours++;
    if(n_j[3] < m && current_grid[n_i[3] * m + n_j[3]] == ALIVE) neighbours++;
    if(n_i[4] < n && n_j[4] < m && current_grid[n_i[4] * m + n_j[4]] == ALIVE) neighbours++;
    if(n_i[5] < n && current_grid[n_i[5] * m + n_j[5]] == ALIVE) neighbours++;
    if(n_i[6] < n && n_j[6] >= 0 && current_grid[n_i[6] * m + n_j[6]] == ALIVE) neighbours++;
    if(n_j[7] >= 0 && current_grid[n_i[7] * m + n_j[7]] == ALIVE) neighbours++;

    if(current_grid[i*m + j] == ALIVE && (neighbours == 2 || neighbours == 3)){
        next_grid[i*m + j] = ALIVE;
    } else if(current_grid[i*m + j] == DEAD && neighbours == 3){
        next_grid[i*m + j] = ALIVE;
    } else {
        next_grid[i*m + j] = DEAD;
    }
}


/*
Implements the game of life on a grid of size `n` times `m`, starting from the `initial_state` configuration.

If `nsteps` is positive, returns the last state reached.
*/
int* gpu_game_of_life(const int *initial_state, int n, int m, int nsteps, float *kernel_time){
    struct timeval start;
    *kernel_time = 0.0;

    unsigned int nThreadsPerBlock, nBlocks;
    int *grid = (int *) malloc(sizeof(int) * n * m);

    int current_step = 0;
    int *tmp = NULL;

    int *dev_grid, *dev_updated_grid;
    CUDA_CHECK_ERROR(hipMalloc(&dev_grid, sizeof(int) * n * m));
    CUDA_CHECK_ERROR(hipMalloc(&dev_updated_grid, sizeof(int) * n * m));
    CUDA_CHECK_ERROR(hipMemcpy(dev_grid, initial_state, sizeof(int) * n * m, hipMemcpyHostToDevice));

    if (n*m > 1024) { 
		nThreadsPerBlock = 1024;
	} else {
		nThreadsPerBlock = n*m;
	}
	nBlocks = (n*m + nThreadsPerBlock - 1) / nThreadsPerBlock;

    while(current_step != nsteps){
        current_step++;

        // Uncomment the following 2 lines if you want to print the state at every step
        // CUDA_CHECK_ERROR(hipMemcpy(grid, dev_grid, sizeof(float) * n * m, hipMemcpyDeviceToHost));
        // visualise(VISUAL_ASCII, current_step, grid, n, m);
        start = init_time();
        gpu_game_of_life_step<<<nBlocks, nThreadsPerBlock>>>(dev_grid, dev_updated_grid, n, m);
        hipDeviceSynchronize();
        *kernel_time += get_elapsed_time(start);
        // swap current and updated grid
        tmp = dev_grid;
        dev_grid = dev_updated_grid;
        dev_updated_grid = tmp;
    }
    // Copy result back to host and free device arrays
    CUDA_CHECK_ERROR(hipMemcpy(grid, dev_grid, sizeof(float) * n * m, hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipFree(dev_grid));
    CUDA_CHECK_ERROR(hipFree(dev_updated_grid));

    return grid;
}

int main(int argc, char **argv)
{


    struct Options *opt = (struct Options *) malloc(sizeof(struct Options));
    getinput(argc, argv, opt);
    int n = opt->n, m = opt->m, nsteps = opt->nsteps;
    int *initial_state = (int *) malloc(sizeof(int) * n * m);
    if(!initial_state){
        printf("Error while allocating memory.\n");
        return -1;
    }
    generate_IC(opt->iictype, initial_state, n, m);
    struct timeval start;
    float kernel_time;

    // Run CPU version
    start = init_time();
    int *cpu_final_state = cpu_game_of_life(initial_state, n, m, nsteps);
    float cpu_elapsed = get_elapsed_time(start);

    // Run GPU version
    start = init_time();
    int *gpu_final_state = gpu_game_of_life(initial_state, n, m, nsteps, &kernel_time);
    float gpu_elapsed = get_elapsed_time(start);


    // Check correctness
    long correct = 0;
    for(int i = 0; i < n; i++) {
        for(int j = 0; j < m; j++) {
            if(cpu_final_state[i*m + j] == gpu_final_state[i*m + j]) {
                correct++;
            }
        }
    }

    printf("CPU:        %f ms\n", cpu_elapsed);
    printf("GPU:        %f ms\n", gpu_elapsed);
    printf("Speedup:    %.2fx\n", cpu_elapsed / gpu_elapsed);
    printf("Kernel:     %f ms\n", kernel_time);
    printf("            %.3f%\n", kernel_time / gpu_elapsed * 100);
    
    // Print number of correct values (with color using escape codes)
    printf("Correct:    ");
    if (correct == n*m) { printf("\033[0;32m");} else { printf("\033[0;31m");}
    printf("%ld/%ld\n", correct, n*m);
    printf("\033[0m");
    
    FILE *fp = fopen("performance.txt", "a");
    fprintf(fp, "%d,%d,%d,%f,%f,%f,%f\n", n, m, nsteps, cpu_elapsed, gpu_elapsed, cpu_elapsed / gpu_elapsed, 
        kernel_time / gpu_elapsed * 100);
    fclose(fp);




        
    free(cpu_final_state);
    free(gpu_final_state);
    free(initial_state);
    free(opt);
    return 0;
}